#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime_api.h>
#include <>
#include "kernel.hpp"

__global__ void CalculateCUDA(int* a, int* b, int dataSize, int calculation){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < dataSize){
		switch(calculation){
		case 0:
			a[idx] += b[idx];
			break;
		case 1:
			a[idx] -= b[idx];
			break;
		case 2:
			a[idx] *= b[idx];
			break;
		case 3:
			a[idx] /= b[idx];
			break;
		default:
			a[idx] = 0;
			break;
		}
	}
}
__global__ void CalculateCUDA(double* a, double* b, int dataSize, int calculation){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < dataSize){
		switch(calculation){
		case 0:
			a[idx] += b[idx];
			break;
		case 1:
			a[idx] -= b[idx];
			break;
		case 2:
			a[idx] *= b[idx];
			break;
		case 3:
			a[idx] /= b[idx];
			break;
		default:
			a[idx] = 0;
			break;
		}
	}
}
__global__ void CalculateCUDA(long double* a, long double* b, int dataSize, int calculation){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < dataSize){
		switch(calculation){
		case 0:
			a[idx] += b[idx];
			break;
		case 1:
			a[idx] -= b[idx];
			break;
		case 2:
			a[idx] *= b[idx];
			break;
		case 3:
			a[idx] /= b[idx];
			break;
		default:
			a[idx] = 0;
			break;
		}
	}
}

__global__ void SigmoidCuda(double* Input){
	Input[0] = 1 / (1+exp((float)-Input[0]));
}
__global__ void SigmoidCuda(long double* Input){
	Input[0] = 1 / (1+exp((float)-Input[0]));
}
__global__ void SigmoidCuda(double* Input, int dataSize){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < dataSize)
		Input[idx] = 1 / (1+exp((float)-Input[idx]));
}
__global__ void SigmoidCuda(long double* Input, int dataSize){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < dataSize)
		Input[idx] = 1 / (1+exp((float)-Input[idx]));
}

__global__ void DSigmoidCuda(double* Input){
	Input[0] = (1 / (1 + exp((float)-Input[0]))) * (1 - (1 / (1 + exp((float)-Input[0]))));
}
__global__ void DSigmoidCuda(long double* Input){
	Input[0] = (1 / (1 + exp((float)-Input[0]))) * (1 - (1 / (1 + exp((float)-Input[0]))));
}
__global__ void DSigmoidCuda(double* Input, int dataSize){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < dataSize)
		Input[idx] = (1 / (1 + exp((float)-Input[idx]))) * (1 - (1 / (1 + exp((float)-Input[idx]))));
}
__global__ void DSigmoidCuda(long double* Input, int dataSize){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < dataSize)
		Input[idx] = (1 / (1 + exp((float)-Input[idx]))) * (1 - (1 / (1 + exp((float)-Input[idx]))));;
}

__global__ void TanHCUDA(double* Input){
	Input[0] = (exp((float)Input[0]) - exp((float)-Input[0])) / (exp((float)Input[0]) + exp((float)-Input[0]));
}
__global__ void TanHCUDA(long double* Input){
	Input[0] = (exp((float)Input[0]) - exp((float)-Input[0])) / (exp((float)Input[0]) + exp((float)-Input[0]));
}
__global__ void TanHCUDA(double* Input, int dataSize){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < dataSize)
		Input[idx] = (exp((float)Input[idx]) - exp((float)-Input[idx])) / (exp((float)Input[idx]) + exp((float)-Input[idx]));
}
__global__ void TanHCUDA(long double* Input, int dataSize){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < dataSize)
		Input[idx] = (exp((float)Input[idx]) - exp((float)-Input[idx])) / (exp((float)Input[idx]) + exp((float)-Input[idx]));
}

__global__ void DTanHCUDA(double* Input){
	Input[0] = 1 - (((exp((float)Input[0]) - exp((float)-Input[0])) / (exp((float)Input[0]) + exp((float)-Input[0]))) * ((exp((float)Input[0]) - exp((float)-Input[0])) / (exp((float)Input[0]) + exp((float)-Input[0]))));
}
__global__ void DTanHCUDA(long double* Input){
	Input[0] = 1 - (((exp((float)Input[0]) - exp((float)-Input[0])) / (exp((float)Input[0]) + exp((float)-Input[0]))) * ((exp((float)Input[0]) - exp((float)-Input[0])) / (exp((float)Input[0]) + exp((float)-Input[0]))));
}
__global__ void DTanHCUDA(double* Input, int dataSize){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < dataSize)
		Input[idx] = 1 - (((exp((float)Input[idx]) - exp((float)-Input[idx])) / (exp((float)Input[idx]) + exp((float)-Input[idx]))) * ((exp((float)Input[idx]) - exp((float)-Input[idx])) / (exp((float)Input[idx]) + exp((float)-Input[idx]))));
}
__global__ void DTanHCUDA(long double* Input, int dataSize){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < dataSize)
		Input[idx] = 1 - (((exp((float)Input[idx]) - exp((float)-Input[idx])) / (exp((float)Input[idx]) + exp((float)-Input[idx]))) * ((exp((float)Input[idx]) - exp((float)-Input[idx])) / (exp((float)Input[idx]) + exp((float)-Input[idx]))));
}

__global__ void MSECUDA(double* Predict, double* Except, int dataSize){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < dataSize){
		Predict[idx] = 0.5 * pow((Except[idx] - Predict[idx]), 2);
	}
}
__global__ void MSECUDA(long double* Predict, long double* Except, int dataSize){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < dataSize){
		Predict[idx] = 0.5 * pow((Except[idx] - Predict[idx]), 2);
	}
}

void Calculate_CUDA(int* a, int* b, int* Output, Kernel_Calculation calculation, int dataSize, int threadSize){
	int* d_a, * d_b;
	if (hipMalloc((void**)&d_a, sizeof(int) * dataSize) != hipSuccess){
		std::cout << "ERROR getting MEMORY from DEVICE!" << std::endl;
		exit(0);
	}
	if(hipMalloc((void**)&d_b, sizeof(int) * dataSize) != hipSuccess){
		std::cout << "ERROR getting MEMORY from DEVICE!" << std::endl;
		hipFree(d_a);
		exit(0);
	}
	if (hipMemcpy(d_a, a, sizeof(int) * dataSize, hipMemcpyHostToDevice) != hipSuccess || hipMemcpy(d_b, b, sizeof(int) * dataSize, hipMemcpyHostToDevice) != hipSuccess){
		std::cout << "ERROR sending DATA to DEVICE!" << std::endl;
		hipFree(d_a); hipFree(d_b);
		exit(0);
	}
	CalculateCUDA<<<dataSize/threadSize + 1, threadSize>>>(d_a, d_b, dataSize, calculation);
	if (hipMemcpy(Output, d_a, sizeof(int) * dataSize, hipMemcpyDeviceToHost) != hipSuccess){
		std::cout << "ERROR sending DATA from DEVICE!" << std::endl;
		hipFree(d_a); hipFree(d_b);
		exit(0);
	}
	hipFree(d_a); hipFree(d_b);
	hipDeviceSynchronize();
}
void Calculate_CUDA(double* a, double* b, double* Output, Kernel_Calculation calculation, int dataSize, int threadSize) {
	double* d_a, * d_b;
	if (hipMalloc((void**)&d_a, sizeof(double) * dataSize) != hipSuccess) {
		std::cout << "ERROR getting MEMORY from DEVICE!" << std::endl;
		exit(0);
	}
	if (hipMalloc((void**)&d_b, sizeof(double) * dataSize) != hipSuccess) {
		std::cout << "ERROR getting MEMORY from DEVICE!" << std::endl;
		hipFree(d_a);
		exit(0);
	}
	if (hipMemcpy(d_a, a, sizeof(double) * dataSize, hipMemcpyHostToDevice) != hipSuccess || hipMemcpy(d_b, b, sizeof(double) * dataSize, hipMemcpyHostToDevice) != hipSuccess) {
		std::cout << "ERROR sending DATA to DEVICE!" << std::endl;
		hipFree(d_a); hipFree(d_b);
		exit(0);
	}
	CalculateCUDA<<<dataSize / threadSize + 1, threadSize>>>(d_a, d_b, dataSize, calculation);
	if (hipMemcpy(Output, d_a, sizeof(double) * dataSize, hipMemcpyDeviceToHost) != hipSuccess) {
		std::cout << "ERROR sending DATA from DEVICE!" << std::endl;
		hipFree(d_a); hipFree(d_b);
		exit(0);
	}
	hipFree(d_a); hipFree(d_b);
	hipDeviceSynchronize();
}
void Calculate_CUDA(long double* a, long double* b, long double* Output, Kernel_Calculation calculation, int dataSize, int threadSize) {
	long double *d_a, *d_b;
	if (hipMalloc((void**)&d_a, sizeof(long double) * dataSize) != hipSuccess) {
		std::cout << "ERROR getting MEMORY from DEVICE!" << std::endl;
		exit(0);
	}
	if (hipMalloc((void**)&d_b, sizeof(long double) * dataSize) != hipSuccess) {
		std::cout << "ERROR getting MEMORY from DEVICE!" << std::endl;
		hipFree(d_a);
		exit(0);
	}
	if (hipMemcpy(d_a, a, sizeof(long double) * dataSize, hipMemcpyHostToDevice) != hipSuccess || hipMemcpy(d_b, b, sizeof(long double) * dataSize, hipMemcpyHostToDevice) != hipSuccess) {
		std::cout << "ERROR sending DATA to DEVICE!" << std::endl;
		hipFree(d_a); hipFree(d_b);
		exit(0);
	}
	CalculateCUDA <<<dataSize / threadSize + 1, threadSize >>>(d_a, d_b, dataSize, calculation);
	if (hipMemcpy(Output, d_a, sizeof(long double) * dataSize, hipMemcpyDeviceToHost) != hipSuccess) {
		std::cout << "ERROR sending DATA from DEVICE!" << std::endl;
		hipFree(d_a); hipFree(d_b);
		exit(0);
	}
	hipFree(d_a); hipFree(d_b);
	hipDeviceSynchronize();
}

double Sigmoid_CUDA(double *Input){
	double *d_Input;
	double Output;
	if (hipMalloc((void**)&d_Input, sizeof(double)) != hipSuccess) {
		std::cout << "ERROR getting MEMORY from DEVICE!" << std::endl;
		return 0;
	}
	if (hipMemcpy(d_Input, Input, sizeof(double), hipMemcpyHostToDevice) != hipSuccess) {
		std::cout << "ERROR sending DATA to DEVICE!" << std::endl;
		hipFree(d_Input);
		return 0;
	}
	SigmoidCuda<<<1, 1>>>(d_Input);
	if (hipMemcpy(&Output, d_Input, sizeof(double), hipMemcpyDeviceToHost) != hipSuccess) {
		std::cout << "ERROR sending DATA from DEVICE!" << std::endl;
		hipFree(d_Input);
		return 0;
	}
	hipFree(d_Input);
	hipDeviceSynchronize();
	return Output;
}
long double Sigmoid_CUDA(long double *Input){
	long double *d_Input;
	long double Output;
	if (hipMalloc((void**)&d_Input, sizeof(long double)) != hipSuccess) {
		std::cout << "ERROR getting MEMORY from DEVICE!" << std::endl;
		return 0;
	}
	if (hipMemcpy(d_Input, Input, sizeof(long double), hipMemcpyHostToDevice) != hipSuccess) {
		std::cout << "ERROR sending DATA to DEVICE!" << std::endl;
		hipFree(d_Input);
		return 0;
	}
	SigmoidCuda<<<1, 1>>>(d_Input);
	if (hipMemcpy(&Output, d_Input, sizeof(long double), hipMemcpyDeviceToHost) != hipSuccess) {
		std::cout << "ERROR sending DATA from DEVICE!" << std::endl;
		hipFree(d_Input);
		return 0;
	}
	hipFree(d_Input);
	hipDeviceSynchronize();
	return Output;
}
void Sigmoid_CUDA(long double *Input, long double *Output, int dataSize, int threadSize){
	long double *d_Input;
	if (hipMalloc((void**)&d_Input, sizeof(long double) * dataSize) != hipSuccess){
		std::cout << "ERROR getting MEMORY from DEVICE!" << std::endl;
		exit(0);
	}
	if (hipMemcpy(d_Input, Input, sizeof(long double) * dataSize, hipMemcpyHostToDevice) != hipSuccess){
		std::cout << "ERROR sending DATA to DEVICE!" << std::endl;
		hipFree(d_Input);
		exit(0);
	}
	SigmoidCuda <<<dataSize / threadSize + 1, threadSize >>>(d_Input, dataSize);
	if (hipMemcpy(Output, d_Input, sizeof(long double) * dataSize, hipMemcpyDeviceToHost) != hipSuccess){
		std::cout << "ERROR sending DATA from DEVICE!" << std::endl;
		hipFree(d_Input);
		exit(0);
	}
	hipFree(d_Input);
	hipDeviceSynchronize();
}
void Sigmoid_CUDA(double *Input, double *Output, int dataSize, int threadSize){
	double *d_Input;
	if (hipMalloc((void**)&d_Input, sizeof(double) * dataSize) != hipSuccess){
		std::cout << "ERROR getting MEMORY from DEVICE!" << std::endl;
		exit(0);
	}
	if (hipMemcpy(d_Input, Input, sizeof(double) * dataSize, hipMemcpyHostToDevice) != hipSuccess){
		std::cout << "ERROR sending DATA to DEVICE!" << std::endl;
		hipFree(d_Input);
		exit(0);
	}
	SigmoidCuda <<<dataSize / threadSize + 1, threadSize >>>(d_Input, dataSize);
	if (hipMemcpy(Output, d_Input, sizeof(double) * dataSize, hipMemcpyDeviceToHost) != hipSuccess){
		std::cout << "ERROR sending DATA from DEVICE!" << std::endl;
		hipFree(d_Input);
		exit(0);
	}
	hipFree(d_Input);
	hipDeviceSynchronize();
}

double DSigmoid_CUDA(double *Input){
	double *d_Input;
	double Output;
	if (hipMalloc((void**)&d_Input, sizeof(double)) != hipSuccess) {
		std::cout << "ERROR getting MEMORY from DEVICE!" << std::endl;
		return 0;
	}
	if (hipMemcpy(d_Input, Input, sizeof(double), hipMemcpyHostToDevice) != hipSuccess) {
		std::cout << "ERROR sending DATA to DEVICE!" << std::endl;
		hipFree(d_Input);
		return 0;
	}
	DSigmoidCuda<<<1, 1>>>(d_Input);
	if (hipMemcpy(&Output, d_Input, sizeof(double), hipMemcpyDeviceToHost) != hipSuccess) {
		std::cout << "ERROR sending DATA from DEVICE!" << std::endl;
		hipFree(d_Input);
		return 0;
	}
	hipFree(d_Input);
	hipDeviceSynchronize();
	return Output;
}
long double DSigmoid_CUDA(long double *Input){
	long double *d_Input;
	long double Output;
	if (hipMalloc((void**)&d_Input, sizeof(long double)) != hipSuccess) {
		std::cout << "ERROR getting MEMORY from DEVICE!" << std::endl;
		return 0;
	}
	if (hipMemcpy(d_Input, Input, sizeof(long double), hipMemcpyHostToDevice) != hipSuccess) {
		std::cout << "ERROR sending DATA to DEVICE!" << std::endl;
		hipFree(d_Input);
		return 0;
	}
	DSigmoidCuda<<<1, 1>>>(d_Input);
	if (hipMemcpy(&Output, d_Input, sizeof(long double), hipMemcpyDeviceToHost) != hipSuccess) {
		std::cout << "ERROR sending DATA from DEVICE!" << std::endl;
		hipFree(d_Input);
		return 0;
	}
	hipFree(d_Input);
	hipDeviceSynchronize();
	return Output;
}
void DSigmoid_CUDA(long double *Input, long double *Output, int dataSize, int threadSize){
	long double *d_Input;
	if (hipMalloc((void**)&d_Input, sizeof(long double) * dataSize) != hipSuccess){
		std::cout << "ERROR getting MEMORY from DEVICE!" << std::endl;
		exit(0);
	}
	if (hipMemcpy(d_Input, Input, sizeof(long double) * dataSize, hipMemcpyHostToDevice) != hipSuccess){
		std::cout << "ERROR sending DATA to DEVICE!" << std::endl;
		hipFree(d_Input);
		exit(0);
	}
	DSigmoidCuda <<<dataSize / threadSize + 1, threadSize >>>(d_Input, dataSize);
	if (hipMemcpy(Output, d_Input, sizeof(long double) * dataSize, hipMemcpyDeviceToHost) != hipSuccess){
		std::cout << "ERROR sending DATA from DEVICE!" << std::endl;
		hipFree(d_Input);
		exit(0);
	}
	hipFree(d_Input);
	hipDeviceSynchronize();
}
void DSigmoid_CUDA(double *Input, double *Output, int dataSize, int threadSize){
	double *d_Input;
	if (hipMalloc((void**)&d_Input, sizeof(double) * dataSize) != hipSuccess){
		std::cout << "ERROR getting MEMORY from DEVICE!" << std::endl;
		exit(0);
	}
	if (hipMemcpy(d_Input, Input, sizeof(double) * dataSize, hipMemcpyHostToDevice) != hipSuccess){
		std::cout << "ERROR sending DATA to DEVICE!" << std::endl;
		hipFree(d_Input);
		exit(0);
	}
	DSigmoidCuda <<<dataSize / threadSize + 1, threadSize >>>(d_Input, dataSize);
	if (hipMemcpy(Output, d_Input, sizeof(double) * dataSize, hipMemcpyDeviceToHost) != hipSuccess){
		std::cout << "ERROR sending DATA from DEVICE!" << std::endl;
		hipFree(d_Input);
		exit(0);
	}
	hipFree(d_Input);
	hipDeviceSynchronize();
}

double TanH_CUDA(double *Input){
	double *d_Input;
	double Output;
	if (hipMalloc((void**)&d_Input, sizeof(double)) != hipSuccess) {
		std::cout << "ERROR getting MEMORY from DEVICE!" << std::endl;
		return 0;
	}
	if (hipMemcpy(d_Input, Input, sizeof(double), hipMemcpyHostToDevice) != hipSuccess) {
		std::cout << "ERROR sending DATA to DEVICE!" << std::endl;
		hipFree(d_Input);
		return 0;
	}
	TanHCUDA<<<1, 1>>>(d_Input);
	if (hipMemcpy(&Output, d_Input, sizeof(double), hipMemcpyDeviceToHost) != hipSuccess) {
		std::cout << "ERROR sending DATA from DEVICE!" << std::endl;
		hipFree(d_Input);
		return 0;
	}
	hipFree(d_Input);
	hipDeviceSynchronize();
	return Output;
}
long double TanH_CUDA(long double *Input){
	long double *d_Input;
	long double Output;
	if (hipMalloc((void**)&d_Input, sizeof(long double)) != hipSuccess) {
		std::cout << "ERROR getting MEMORY from DEVICE!" << std::endl;
		return 0;
	}
	if (hipMemcpy(d_Input, Input, sizeof(long double), hipMemcpyHostToDevice) != hipSuccess) {
		std::cout << "ERROR sending DATA to DEVICE!" << std::endl;
		hipFree(d_Input);
		return 0;
	}
	TanHCUDA<<<1, 1>>>(d_Input);
	if (hipMemcpy(&Output, d_Input, sizeof(long double), hipMemcpyDeviceToHost) != hipSuccess) {
		std::cout << "ERROR sending DATA from DEVICE!" << std::endl;
		hipFree(d_Input);
		return 0;
	}
	hipFree(d_Input);
	hipDeviceSynchronize();
	return Output;
}
void TanH_CUDA(long double *Input, long double *Output, int dataSize, int threadSize){
	long double *d_Input;
	if (hipMalloc((void**)&d_Input, sizeof(long double) * dataSize) != hipSuccess){
		std::cout << "ERROR getting MEMORY from DEVICE!" << std::endl;
		exit(0);
	}
	if (hipMemcpy(d_Input, Input, sizeof(long double) * dataSize, hipMemcpyHostToDevice) != hipSuccess){
		std::cout << "ERROR sending DATA to DEVICE!" << std::endl;
		hipFree(d_Input);
		exit(0);
	}
	TanHCUDA <<<dataSize / threadSize + 1, threadSize >>>(d_Input, dataSize);
	if (hipMemcpy(Output, d_Input, sizeof(long double) * dataSize, hipMemcpyDeviceToHost) != hipSuccess){
		std::cout << "ERROR sending DATA from DEVICE!" << std::endl;
		hipFree(d_Input);
		exit(0);
	}
	hipFree(d_Input);
	hipDeviceSynchronize();
}
void TanH_CUDA(double *Input, double *Output, int dataSize, int threadSize){
	double *d_Input;
	if (hipMalloc((void**)&d_Input, sizeof(double) * dataSize) != hipSuccess){
		std::cout << "ERROR getting MEMORY from DEVICE!" << std::endl;
		exit(0);
	}
	if (hipMemcpy(d_Input, Input, sizeof(double) * dataSize, hipMemcpyHostToDevice) != hipSuccess){
		std::cout << "ERROR sending DATA to DEVICE!" << std::endl;
		hipFree(d_Input);
		exit(0);
	}
	TanHCUDA <<<dataSize / threadSize + 1, threadSize >>>(d_Input, dataSize);
	if (hipMemcpy(Output, d_Input, sizeof(double) * dataSize, hipMemcpyDeviceToHost) != hipSuccess){
		std::cout << "ERROR sending DATA from DEVICE!" << std::endl;
		hipFree(d_Input);
		exit(0);
	}
	hipFree(d_Input);
	hipDeviceSynchronize();
}

double DTanH_CUDA(double *Input){
	double *d_Input;
	double Output;
	if (hipMalloc((void**)&d_Input, sizeof(double)) != hipSuccess) {
		std::cout << "ERROR getting MEMORY from DEVICE!" << std::endl;
		return 0;
	}
	if (hipMemcpy(d_Input, Input, sizeof(double), hipMemcpyHostToDevice) != hipSuccess) {
		std::cout << "ERROR sending DATA to DEVICE!" << std::endl;
		hipFree(d_Input);
		return 0;
	}
	DTanHCUDA<<<1, 1>>>(d_Input);
	if (hipMemcpy(&Output, d_Input, sizeof(double), hipMemcpyDeviceToHost) != hipSuccess) {
		std::cout << "ERROR sending DATA from DEVICE!" << std::endl;
		hipFree(d_Input);
		return 0;
	}
	hipFree(d_Input);
	hipDeviceSynchronize();
	return Output;
}
long double DTanH_CUDA(long double *Input){
	long double *d_Input;
	long double Output;
	if (hipMalloc((void**)&d_Input, sizeof(long double)) != hipSuccess) {
		std::cout << "ERROR getting MEMORY from DEVICE!" << std::endl;
		return 0;
	}
	if (hipMemcpy(d_Input, Input, sizeof(long double), hipMemcpyHostToDevice) != hipSuccess) {
		std::cout << "ERROR sending DATA to DEVICE!" << std::endl;
		hipFree(d_Input);
		return 0;
	}
	DTanHCUDA<<<1, 1>>>(d_Input);
	if (hipMemcpy(&Output, d_Input, sizeof(long double), hipMemcpyDeviceToHost) != hipSuccess) {
		std::cout << "ERROR sending DATA from DEVICE!" << std::endl;
		hipFree(d_Input);
		return 0;
	}
	hipFree(d_Input);
	hipDeviceSynchronize();
	return Output;
}
void DTanH_CUDA(long double *Input, long double *Output, int dataSize, int threadSize){
	long double *d_Input;
	if (hipMalloc((void**)&d_Input, sizeof(long double) * dataSize) != hipSuccess){
		std::cout << "ERROR getting MEMORY from DEVICE!" << std::endl;
		exit(0);
	}
	if (hipMemcpy(d_Input, Input, sizeof(long double) * dataSize, hipMemcpyHostToDevice) != hipSuccess){
		std::cout << "ERROR sending DATA to DEVICE!" << std::endl;
		hipFree(d_Input);
		exit(0);
	}
	DTanHCUDA <<<dataSize / threadSize + 1, threadSize >>>(d_Input, dataSize);
	if (hipMemcpy(Output, d_Input, sizeof(long double) * dataSize, hipMemcpyDeviceToHost) != hipSuccess){
		std::cout << "ERROR sending DATA from DEVICE!" << std::endl;
		hipFree(d_Input);
		exit(0);
	}
	hipFree(d_Input);
	hipDeviceSynchronize();
}
void DTanH_CUDA(double *Input, double *Output, int dataSize, int threadSize){
	double *d_Input;
	if (hipMalloc((void**)&d_Input, sizeof(double) * dataSize) != hipSuccess){
		std::cout << "ERROR getting MEMORY from DEVICE!" << std::endl;
		exit(0);
	}
	if (hipMemcpy(d_Input, Input, sizeof(double) * dataSize, hipMemcpyHostToDevice) != hipSuccess){
		std::cout << "ERROR sending DATA to DEVICE!" << std::endl;
		hipFree(d_Input);
		exit(0);
	}
	DTanHCUDA <<<dataSize / threadSize + 1, threadSize >>>(d_Input, dataSize);
	if (hipMemcpy(Output, d_Input, sizeof(double) * dataSize, hipMemcpyDeviceToHost) != hipSuccess){
		std::cout << "ERROR sending DATA from DEVICE!" << std::endl;
		hipFree(d_Input);
		exit(0);
	}
	hipFree(d_Input);
	hipDeviceSynchronize();
}

void MSE_CUDA(long double *Predicted, long double *Expected, long double *Output, int dataSize, int threadSize){
	long double* d_Predicted, * d_Expected;
	if (hipMalloc((void**)&d_Predicted, sizeof(long double) * dataSize) != hipSuccess){
		std::cout << "ERROR getting MEMORY from DEVICE!" << std::endl;
		exit(0);
	}
	if(hipMalloc((void**)&d_Expected, sizeof(long double) * dataSize) != hipSuccess){
		std::cout << "ERROR getting MEMORY from DEVICE!" << std::endl;
		hipFree(d_Predicted);
		exit(0);
	}
	if (hipMemcpy(d_Predicted, Predicted, sizeof(long double) * dataSize, hipMemcpyHostToDevice) != hipSuccess || hipMemcpy(d_Expected, Expected, sizeof(long double) * dataSize, hipMemcpyHostToDevice) != hipSuccess){
		std::cout << "ERROR sending DATA to DEVICE!" << std::endl;
		hipFree(d_Predicted); hipFree(d_Expected);
		exit(0);
	}
	MSECUDA<<<dataSize/threadSize + 1, threadSize>>>(d_Predicted, d_Expected, dataSize);
	if (hipMemcpy(Output, d_Predicted, sizeof(long double) * dataSize, hipMemcpyDeviceToHost) != hipSuccess){
		std::cout << "ERROR sending DATA from DEVICE!" << std::endl;
		hipFree(d_Predicted); hipFree(d_Expected);
		exit(0);
	}
	hipFree(d_Predicted); hipFree(d_Expected);
	hipDeviceSynchronize();
}
void MSE_CUDA(double *Predicted, double *Expected, double *Output, int dataSize, int threadSize){
	double* d_Predicted, * d_Expected;
	if (hipMalloc((void**)&d_Predicted, sizeof(double) * dataSize) != hipSuccess){
		std::cout << "ERROR getting MEMORY from DEVICE!" << std::endl;
		exit(0);
	}
	if(hipMalloc((void**)&d_Expected, sizeof(double) * dataSize) != hipSuccess){
		std::cout << "ERROR getting MEMORY from DEVICE!" << std::endl;
		hipFree(d_Predicted);
		exit(0);
	}
	if (hipMemcpy(d_Predicted, Predicted, sizeof(double) * dataSize, hipMemcpyHostToDevice) != hipSuccess || hipMemcpy(d_Expected, Expected, sizeof(double) * dataSize, hipMemcpyHostToDevice) != hipSuccess){
		std::cout << "ERROR sending DATA to DEVICE!" << std::endl;
		hipFree(d_Predicted); hipFree(d_Expected);
		exit(0);
	}
	MSECUDA<<<dataSize/threadSize + 1, threadSize>>>(d_Predicted, d_Expected, dataSize);
	if (hipMemcpy(Output, d_Predicted, sizeof(double) * dataSize, hipMemcpyDeviceToHost) != hipSuccess){
		std::cout << "ERROR sending DATA from DEVICE!" << std::endl;
		hipFree(d_Predicted); hipFree(d_Expected);
		exit(0);
	}
	hipFree(d_Predicted); hipFree(d_Expected);
	hipDeviceSynchronize();
}

void _BasicKernel_Cuda_RESET(){
	hipDeviceReset();
}