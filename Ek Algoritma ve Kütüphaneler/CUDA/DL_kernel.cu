#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime_api.h>
#include <>
#include "DL_kernel.hpp"

__device__ long double DL_Sigmoid(long double i){
	return 1 / (1 + exp((float)-i));
}
__device__ long double DL_TanH(long double i){
	return (exp((float)i) - exp((float)-i)) / (exp((float)i) + exp((float)-i));
}
__device__ long double DL_DTanH(long double i) {
    return 1 - (((exp((float)i) - exp((float)-i)) / (exp((float)i) + exp((float)-i))) * ((exp((float)i) - exp((float)-i)) / (exp((float)i) + exp((float)-i))));
}

__global__ void LSTM_FEEDFORWARD(long double *d_Neurons,long double *d_Input,long double *d_STM,long double *d_LTM,long double *d_IW,long double *d_STMW,long double *d_BW){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < 4){
		switch(idx){
		case 2:
			d_Neurons[idx] = DL_TanH((d_Input[0] * d_IW[idx]) + (d_STM[0] * d_STMW[idx]) + d_BW[idx]);
			break;
		default:
			d_Neurons[idx] = DL_Sigmoid((d_Input[0] * d_IW[idx]) + (d_STM[0] * d_STMW[idx]) + d_BW[idx]);
			break;
		}
	}
}
__global__ void LSTM_FEEDFORWARD_2(long double *d_Neurons,long double *d_LTM, long double *d_oldLTM,long double *d_STM,long double *d_oldSTM){
	d_oldLTM[0] = d_LTM[0];
	d_LTM[0] *= d_Neurons[0];
	d_LTM[0] += d_Neurons[1] * d_Neurons[2];
	d_Neurons[4] = DL_TanH(d_LTM[0]);
	d_oldSTM[0] = d_STM[0];
	d_STM[0] = d_Neurons[3] * d_Neurons[4];
}
__global__ void LSTM_BACKPROPAGATION(long double *d_Neurons,long double *d_Loss,long double *d_STM,long double *d_NINPUT,long double *d_LTM, long double *d_NSD){
	d_Loss[0] = d_STM[0] - d_NINPUT[0];
	d_NSD[0] = d_Loss[0] * d_Neurons[3] * DL_DTanH(d_LTM[0]);
}
__global__ void LSTM_BACKPROPAGATION_2(long double *d_Neurons,long double *d_FGD,long double *d_IGD,long double *d_CGD,long double *d_NSD,long double* d_oldLTM){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < 3){
		switch(idx){
		case 0:
			d_FGD[0] = d_NSD[0] * d_oldLTM[0] * d_Neurons[0] * (1 - d_Neurons[0]);
			break;
		case 1:
			d_IGD[0] = d_NSD[0] * d_Neurons[2] * d_Neurons[1] * (1 - d_Neurons[1]);
			break;
		case 2:
			d_CGD[0] = d_NSD[0] * d_Neurons[1] * (1 - (d_Neurons[2] * d_Neurons[2]));
			break;
		}
	}
}
__global__ void LSTM_UPDATE_WEIGHTS(long double *FGD, long double *IGD, long double *CGD, long double *IW, long double *STMW, long double *BW, long double *oldSTM, long double *learningRate, long double *Input){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < 4){
		switch(idx){
		case 0:
			IW[idx] -= FGD[0] * Input[0] * learningRate[0];
            STMW[idx] -= FGD[0] * oldSTM[0] * learningRate[0];
            BW[idx] -= FGD[0] * learningRate[0];
			break;
		case 1:
			IW[idx] -= IGD[0] * Input[0] * learningRate[0];
            STMW[idx] -= IGD[0] * oldSTM[0] * learningRate[0];
            BW[idx] -= IGD[0] * learningRate[0];
			break;
		default:
			IW[idx] -= CGD[0] * Input[0] * learningRate[0];
            STMW[idx] -= CGD[0] * oldSTM[0] * learningRate[0];
            BW[idx] -= CGD[0] * learningRate[0];
			break;
		}
	}
}
__global__ void NN_FEEDFORWARD(long double *d_firstValue,long double *d_secondLayer,long double *d_Weights, int layerSize){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < layerSize){
		d_secondLayer[idx] += d_firstValue[0] * d_Weights[idx];
	}
}
void CUDA_LSTM_BEGIN(long double *IW, long double *STMW, long double *BW,
			long double **d_LR, long double **d_LTM, long double **d_STM, long double **d_Neurons, long double **d_Loss,
            long double **d_FGD, long double **d_IGD, long double **d_CGD, long double **d_Input, long double **d_IW,
			long double **d_STMW, long double **d_BW, long double **d_oldSTM, long double **d_oldLTM, long double **d_NSD,
			long double **d_NINPUT){
	hipError_t err;
	err = hipMalloc((void**)d_FGD, sizeof(long double));
	if(err != hipSuccess){
		std::cout << "CUDA CREATE VARIABLE FGD!" << std::endl;
		std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
		exit(0);
	}
	err = hipMalloc((void**)d_IGD, sizeof(long double));
	if(err != hipSuccess){
		std::cout << "CUDA CREATE VARIABLE IGD!" << std::endl;
		std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
		exit(0);
	}
	err = hipMalloc((void**)d_CGD, sizeof(long double));
	if(err != hipSuccess){
		std::cout << "CUDA CREATE VARIABLE CGD!" << std::endl;
		std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
		exit(0);
	}
	err = hipMalloc((void**)d_IW, sizeof(long double) * 4);
	if(err != hipSuccess){
		std::cout << "CUDA CREATE VARIABLE IW!" << std::endl;
		std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
		exit(0);
	}
	err = hipMalloc((void**)d_STMW, sizeof(long double) * 4);
	if(err != hipSuccess){
		std::cout << "CUDA CREATE VARIABLE STMW!" << std::endl;
		std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
		exit(0);
	}
	err = hipMalloc((void**)d_BW, sizeof(long double) * 4);
	if(err != hipSuccess){
		std::cout << "CUDA CREATE VARIABLE BW!" << std::endl;
		std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
		exit(0);
	}
	err = hipMalloc((void**)d_oldSTM, sizeof(long double));
	if(err != hipSuccess){
		std::cout << "CUDA CREATE VARIABLE OLDSTM!" << std::endl;
		std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
		exit(0);
	}
	err = hipMalloc((void**)d_LR, sizeof(long double));
	if(err != hipSuccess){
		std::cout << "CUDA CREATE VARIABLE LR!" << std::endl;
		std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
		exit(0);
	}
	err = hipMalloc((void**)d_Input, sizeof(long double));
	if(err != hipSuccess){
		std::cout << "CUDA CREATE VARIABLE INPUT!" << std::endl;
		std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
		exit(0);
	}
	err = hipMalloc((void**)d_LTM, sizeof(long double));
	if(err != hipSuccess){
		std::cout << "CUDA CREATE VARIABLE LTM!" << std::endl;
		std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
		exit(0);
	}
	err = hipMalloc((void**)d_STM, sizeof(long double));
	if(err != hipSuccess){
		std::cout << "CUDA CREATE VARIABLE STM!" << std::endl;
		std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
		exit(0);
	}
	err = hipMalloc((void**)d_Neurons, sizeof(long double) * 5);
	if(err != hipSuccess){
		std::cout << "CUDA CREATE VARIABLE NEURONS!" << std::endl;
		std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
		exit(0);
	}
	err = hipMalloc((void**)d_Loss, sizeof(long double));
	if(err != hipSuccess){
		std::cout << "CUDA CREATE VARIABLE LOSS!" << std::endl;
		std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
		exit(0);
	}
	err = hipMalloc((void**)d_oldLTM, sizeof(long double));
	if(err != hipSuccess){
		std::cout << "CUDA CREATE VARIABLE OLDLTM!" << std::endl;
		std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
		exit(0);
	}
	err = hipMalloc((void**)d_NSD, sizeof(long double));
	if(err != hipSuccess){
		std::cout << "CUDA CREATE VARIABLE NSD!" << std::endl;
		std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
		exit(0);
	}
	err = hipMalloc((void**)d_NINPUT, sizeof(long double));
	if(err != hipSuccess){
		std::cout << "CUDA CREATE VARIABLE NINPUT!" << std::endl;
		std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
		exit(0);
	}
	err = hipMemcpy(*d_IW, IW, sizeof(long double) * 4, hipMemcpyHostToDevice);
	if(err != hipSuccess){
		std::cout << "CUDA SAVE VALUE IW!" << std::endl;
		std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
		exit(0);
	}
	err = hipMemcpy(*d_STMW, STMW, sizeof(long double) * 4, hipMemcpyHostToDevice);
	if(err != hipSuccess){
		std::cout << "CUDA SAVE VALUE STMW!" << std::endl;
		std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
		exit(0);
	}
	err = hipMemcpy(*d_BW, BW, sizeof(long double) * 4, hipMemcpyHostToDevice);
	if(err != hipSuccess){
		std::cout << "CUDA SAVE VALUE BW!" << std::endl;
		std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
		exit(0);
	}
}
void CUDA_LSTM_FEEDFORWARD(long double *Input, long double *NInput, long double *d_NINPUT, long double *d_Neurons,long double *d_Input,long double *d_STM,long double *d_LTM,long double *d_IW,long double *d_STMW,long double *d_BW,long double *d_oldSTM,long double *d_oldLTM, int threadSize){
	hipError_t err = hipMemcpy(d_Input, Input, sizeof(long double), hipMemcpyHostToDevice);
	if(err != hipSuccess){
		std::cout << "CUDA LSTM SAVE INPUT!" << std::endl;
		std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
		exit(0);
	}
	err = hipMemcpy(d_NINPUT, NInput, sizeof(long double), hipMemcpyHostToDevice);
	if(err != hipSuccess){
		std::cout << "CUDA LSTM SAVE NEXT INPUT!" << std::endl;
		std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
		exit(0);
	}
	LSTM_FEEDFORWARD<<<(4+threadSize -1) / threadSize, threadSize>>>(d_Neurons, d_Input, d_STM, d_LTM, d_IW, d_STMW, d_BW);
	hipDeviceSynchronize();
	LSTM_FEEDFORWARD_2<<<1, 1>>>(d_Neurons, d_LTM, d_oldLTM, d_STM, d_oldSTM);
	hipDeviceSynchronize();
}
void CUDA_LSTM_BACKPROPAGATION(long double *d_Neurons, long double *d_oldLTM, long double *d_LTM, long double *d_STM, long double *d_Loss,long double *d_NSD,long double *d_FGD,long double *d_IGD,long double *d_CGD,long double* d_IW,long double* d_STMW,long double* d_BW,long double *d_oldSTM,long double *d_LR,long double *d_Input, long double *d_NINPUT, long double *learningRate, int threadSize){
	hipError_t err = hipMemcpy(d_LR, learningRate, sizeof(long double), hipMemcpyHostToDevice);
	if(err != hipSuccess){
		std::cout << "CUDA LSTM SAVE LEARNING RATE!" << std::endl;
		std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
		exit(0);
	}
	LSTM_BACKPROPAGATION<<<1, 1>>>(d_Neurons, d_Loss, d_STM, d_NINPUT, d_LTM, d_NSD);
	hipDeviceSynchronize();
	LSTM_BACKPROPAGATION_2<<<(3+threadSize - 1) / threadSize, threadSize>>>(d_Neurons, d_FGD, d_IGD, d_CGD, d_NSD, d_oldLTM);
	hipDeviceSynchronize();
	LSTM_UPDATE_WEIGHTS<<<(4+threadSize - 1)/threadSize, threadSize>>>(d_FGD, d_IGD, d_CGD, d_IW, d_STMW, d_BW, d_oldSTM, d_LR, d_Input);
	hipDeviceSynchronize();
}
void CUDA_LSTM_SAVE_LAST_WEIGHTS(long double *InputWeights,long double *d_IW,long double *STMWeights, long double *d_STMW, long double *BiaS,long double *d_BW){
	hipError_t err = hipMemcpy(InputWeights,d_IW,sizeof(long double)*4,hipMemcpyDeviceToHost);
	if(err != hipSuccess){
		std::cout << "CUDA LSTM SAVE LAST INPUT WEIHGTS!" << std::endl;
		std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
		exit(0);
	}
	err = hipMemcpy(STMWeights,d_STMW,sizeof(long double)*4,hipMemcpyDeviceToHost);
	if(err != hipSuccess){
		std::cout << "CUDA LSTM SAVE LAST STM WEIGHTS!" << std::endl;
		std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
		exit(0);
	}
	err = hipMemcpy(BiaS,d_BW,sizeof(long double)*4,hipMemcpyDeviceToHost);
	if(err != hipSuccess){
		std::cout << "CUDA LSTM SAVE LAST BIASW!" << std::endl;
		std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
		exit(0);
	}
}
void CUDA_LSTM_RESET_LTM_STM(long double *d_LTM,long double *d_STM){
	long double ltm_stm = 0;
	hipError_t err = hipMemcpy(d_LTM, &ltm_stm, sizeof(long double), hipMemcpyHostToDevice);
	if(err != hipSuccess){
		std::cout << "CUDA LSTM RESET LTM!" << std::endl;
		std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
		exit(0);
	}
	err = hipMemcpy(d_STM, &ltm_stm, sizeof(long double), hipMemcpyHostToDevice);
	if(err != hipSuccess){
		std::cout << "CUDA LSTM RESET STM!" << std::endl;
		std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
		exit(0);
	}
	hipDeviceSynchronize();
}
void CUDA_NN_FEEDFORWARD(long double *firstValue,long double *secondLayer,long double *Weights,int secondLayerSize, int threadSize){
	long double *d_firstValue, *d_secondLayer, *d_Weights;
	hipError_t err = hipMalloc((void**)&d_firstValue, sizeof(long double));
	if(err != hipSuccess){
		std::cout << "CUDA NN FEEDFORWARD GET MEMORY FIRST VALUE!" << std::endl;
		std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
		exit(0);
	}
	err = hipMalloc((void**)&d_secondLayer, sizeof(long double) * secondLayerSize);
	if(err != hipSuccess){
		std::cout << "CUDA NN FEEDFORWARD GET MEMORY SECOND LAYER!" << std::endl;
		std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
		exit(0);
	}
	err = hipMalloc((void**)&d_Weights, sizeof(long double) * secondLayerSize);
	if(err != hipSuccess){
		std::cout << "CUDA NN FEEDFORWARD GET MEMORY WEIGHTS!" << std::endl;
		std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
		exit(0);
	}
	err = hipMemcpy(d_firstValue, firstValue, sizeof(long double), hipMemcpyHostToDevice);
	if(err != hipSuccess){
		std::cout << "CUDA NN FEEDFORWARD COPY FIRST VALUE!" << std::endl;
		std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
		exit(0);
	}
	err =hipMemcpy(d_Weights, Weights, sizeof(long double) * secondLayerSize, hipMemcpyHostToDevice);
	if(err != hipSuccess){
		std::cout << "CUDA NN FEEDFORWARD COPY WEIGHTS!" << std::endl;
		std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
		exit(0);
	}
	err =hipMemcpy(d_secondLayer, secondLayer, sizeof(long double) * secondLayerSize, hipMemcpyHostToDevice);
	if(err != hipSuccess){
		std::cout << "CUDA NN FEEDFORWARD COPY WEIGHTS!" << std::endl;
		std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
		exit(0);
	}
	NN_FEEDFORWARD<<<(secondLayerSize + threadSize -1) / threadSize, threadSize>>>(d_firstValue, d_secondLayer, d_Weights, secondLayerSize);
	err =hipMemcpy(secondLayer, d_secondLayer, sizeof(long double) * secondLayerSize, hipMemcpyDeviceToHost);
	if(err != hipSuccess){
		std::cout << "CUDA NN FEEDFORWARD COPY WEIGHTS!" << std::endl;
		std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
		exit(0);
	}
	hipFree(d_firstValue); hipFree(d_secondLayer); hipFree(d_Weights);
	hipDeviceSynchronize();
}
long double _getValueCUDA_(long double* Variable){
	long double Output;
	if (Variable == NULL) {
		std::cerr << "Error: Variable is null!" << std::endl;
	}
	hipError_t err = hipMemcpy(&Output, Variable, sizeof(long double), hipMemcpyDeviceToHost);
	if(err != hipSuccess){
		std::cout << "CUDA GET VALUE!" << std::endl;
		std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
		exit(0);
	}
	return Output;
}
void _DLkernel_Cuda_RESET(){
	hipDeviceReset();
}